#include "hip/hip_runtime.h"
/*
 * Title: 2D Image Convolution on GPU by using Shared Memory and Constant Memory.
 *
 * Image Size: 2048 X 2048
 * Mask Size: 64 X 64
 * TILE_WIDTH 32
 *
 *
 * */
#include<stdio.h>
#include<hip/hip_runtime_api.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<assert.h>
#include<base.h>
#include <time.h>
#define Mask_width  3
#define Mask_height 3
#define Mask_radius_x Mask_width/2
#define Mask_radius_y Mask_height/2
#define TILE_WIDTH 32 //16 X 16 TILE
#define B_x (TILE_WIDTH + Mask_width - 1)
#define B_y (TILE_WIDTH + Mask_height - 1)
#define clamp(x) (max(max((x), 0.0),x))
#define SIZE 224
#define max4(w,x,y,z) max(max(max(w,x),y),z)


typedef enum
{
  CONV_1 = 512,
  CONV_2 = 2,
  CONV_3 = 3,
}ch;
const int out = ch(CONV_1);
float *dense_1 = (float*)(malloc(dense[0][0]*dense[0][1] * sizeof(float)));
float *dense_2 = (float*)(malloc(dense[1][0]*dense[1][1] * sizeof(float)));
float *dense_3 = (float*)(malloc(dense[2][0]*dense[2][1] * sizeof(float)));
float *bias_1 = (float*)(malloc(dense[0][1] * sizeof(float)));
float *bias_2 = (float*)(malloc(dense[1][1] * sizeof(float)));
float *bias_3 = (float*)(malloc(dense[2][1] * sizeof(float)));
// Supporting functions go here

// Function to convert input image(.txt) to normalized BGR format

void softmax(float *out, int sh_out) {
	int i;
	float max_val, sum;
	max_val = out[0];
	for (i = 1; i < sh_out; i++) {
		if (out[i] > max_val)
			max_val = out[i];
	}
	sum = 0.0;
	for (i = 0; i < sh_out; i++) {
		out[i] = exp(out[i] - max_val);
		sum += out[i];
	}
	for (i = 0; i < sh_out; i++) {
		out[i] /= sum;
	}
}

//@@ INSERT CODE HERE
__global__ void fully_connected(float *I, const float* __restrict__ M, float *P,int channels, int width, int height,int outputChannels)
{
   __shared__ float F_ds[7][7];
   //__shared__ float W_ds[f_y][f_x];
   int k;

   //float acc[5] ={0};
   float acc[4096] ={0};

     for (k = 0; k < channels; k++)
     {

      F_ds[threadIdx.x][threadIdx.y] = I[(threadIdx.x + (blockIdx.x * TILE_WIDTH)) * (width * channels) + threadIdx.y * channels + k + blockIdx.y * (TILE_WIDTH) ];

        __syncthreads();

        int y, x,z;
        if(threadIdx.y == 0 && threadIdx.x ==0 &&blockIdx.y == 0 && blockIdx.x == 0)
        {

          for(int z =0;z<outputChannels;z++)
          {
              for(int i =0; i< TILE_WIDTH; i++)
              {
                for(int j =0; j<TILE_WIDTH; j++)
                {
                     //printf("%.2f \t %.2f \t",F_ds[i][j],M[i*TILE_WIDTH + TILE_WIDTH*TILE_WIDTH*z+ outputChannels*k*TILE_WIDTH*TILE_WIDTH+ j]);

                    acc[z] += F_ds[i][j] * M[i*TILE_WIDTH + TILE_WIDTH*TILE_WIDTH*z+ outputChannels*k*TILE_WIDTH*TILE_WIDTH+ j];
                    //acc[z] += F_ds[i][j] *M[j*outputChannels+outputChannels+(i*7)*outputChannels + k * 7 * 7 * outputChannels];

                }

              //  printf("\n");
              }
            //  printf("done for z:%d and k:%d\n",z,k);
          }


       }
       __syncthreads();
       //printf("done\n");

   }
   if(threadIdx.y == 0 && threadIdx.x ==0 &&blockIdx.y == 0 && blockIdx.x == 0)
   {
     for(int z =0;z<outputChannels;z++)
     {
        //printf("%.2f\t",acc[z]);
        P[z] = acc[z];
      }

   }


}
void normalizeBGR(float *hostInputImageData)
{
    float coef[3] = { 103.939, 116.779, 123.68 };
	//float *hostInputImageData = (float *) malloc(sizeof(float)*SIZE*SIZE*INPUT_CHANNELS);
	FILE *input = fopen("vol.txt", "r");
	float dval, val;
	int count =0;
    for(int i=0;i<SIZE*SIZE*INPUT_CHANNELS;i++)
    {
	    fscanf(input, "%f", &dval);
		//int n = (i+1);
	    if(count == 0)
		    val = dval - coef[count];
		if(count == 1)
		    val = dval - coef[count];
		if(count == 2)
		    val = dval -  coef[count];
		hostInputImageData[i]= val;
        count++;
        if(count == 3)
            count = 0;
    }
	FILE *results = fopen("results.txt", "w");
	for(int i=0;i<SIZE*SIZE*INPUT_CHANNELS;i++) {
				if(i % (SIZE*INPUT_CHANNELS) == 0 && i != 0)
					fprintf(results, "\n");
				fprintf(results, "%.5f\t",hostInputImageData[i]);
			}

	fclose(input);
	fclose(results);
}
static unsigned int pos = 0;
FILE *weight = fopen("weights.txt", "r");

// Read the weights from weights.txt file and store in memory
// Read the weights from weights.txt file and store in memory
void readWeights(int level,float *wconv, float *bias){
	float dval;
	//int i, j, k, l, z;
	FILE *weight;
	FILE *conv;
    // Read the weights from text file
	weight = fopen("weights.txt", "r");
	conv = fopen("conv.txt", "w");
	if (weight == NULL) {
		printf("File weights absent\n");
		exit(1);
	}
//  int skip = level;
	// Memory allocation
	//float *wconv = (float *) malloc(sizeof(float)*layers[level][0]*layers[level][1]*layers[level][2]*layers[level][3]);
  if(level != 0)
  {
    for(int s =0; s<level; s++)
    {
      for(int j=1; j<=layers[s][0]*layers[s][1]; j++)
    	{
    		for(int k =0; k<CONV_SIZE * CONV_SIZE; k++)
    		{
    			fscanf(weight, "%f", &dval);
    			//*(wconv + j*CONV_SIZE * CONV_SIZE - 1 -k) = dval;
    		}
     	}

      for (int i = 0; i < layers[s][0]; i++) {
        fscanf(weight, "%f", &dval);
      }
    }
  }


	for(int j=1; j<=layers[level][0]*layers[level][1]; j++)
	{
		for(int k =0; k<CONV_SIZE * CONV_SIZE; k++)
		{
			fscanf(weight, "%f", &dval);
			*(wconv + j*CONV_SIZE * CONV_SIZE - 1 -k) = dval;
		}
	}

	for (int i = 0; i < layers[level][0]*layers[level][1]*layers[level][2]*layers[level][3]; i++) {
		fprintf(conv, "%.5f\t",wconv[i]);
	}

	FILE *bias1 = fopen("bias.txt", "w");
	// for (int i = 0; i < layers[level][0]; i++) {
	// 	fscanf(weight, "%f", &dval);
	// 	bias[i] = dval;
	// 	fprintf(bias1, "%.5f\t",bias[i]);
	// }

  int i =0;
  while(i<layers[level][0])// && fscanf(weight, "%f", &dval) != EOF)
  {
  	fscanf(weight, "%f", &dval);
    bias[i] = dval;
    fprintf(bias1, "%.5f\t",bias[i]);
    i++;
  }

  if(level == 12)
  {
    int i,j;
  		printf("Read dense block %d weights\n", 0);
  		for (i = 0; i < dense[0][0]; i++) {
  			for (j = 0; j < dense[0][1]; j++) {
  				fscanf(weight, "%f", &dval);
  				*(dense_1 + i)= dval;
  			}
  		}
  		for (i = 0; i < dense[0][1]; i++) {
  			fscanf(weight, "%f", &dval);
  			*(bias_1+i) = dval;
  		}
      printf("Read dense block %d weights\n", 1);
      for (i = 0; i < dense[1][0]; i++) {
        for (j = 0; j < dense[1][1]; j++) {
          fscanf(weight, "%f", &dval);
          *(dense_2 + i)= dval;
        }
      }
      for (i = 0; i < dense[1][1]; i++) {
        fscanf(weight, "%f", &dval);
        *(bias_2+i) = dval;
      }
      printf("Read dense block %d weights\n", 2);
      for (i = 0; i < dense[2][0]; i++) {
        for (j = 0; j < dense[2][1]; j++) {
          fscanf(weight, "%f", &dval);
          *(dense_3 + i)= dval;
        }
      }
      for (i = 0; i < dense[2][1]; i++) {
        fscanf(weight, "%f", &dval);
        *(bias_3+i) = dval;
      }

  }
  //pos = ftell(weight);

  fclose(weight);
	fclose(bias1);
	fclose(conv);
}
void dense_weights(int level,float *wconv, float *bias)
{
  printf("Read justn");
  fseek(weight,pos,0);
  int z =level;
  float dval;
  // Reading dense weights
	//for (z = 0; z < 3; z++) {
		printf("Read dense block %d weights\n", z);
		for (int i = 0; i < dense[z][0]*dense[z][1]; i++) {
				fscanf(weight, "%f", &dval);
				*(wconv+i) = dval;
			}
	//	}
		for (int i = 0; i < dense[z][1]; i++) {
			fscanf(weight, "%f", &dval);
			*(bias+i) = dval;
		}
	//}
  pos = ftell(weight);
  fclose(weight);

}

__global__ void maxpool(float *image, float * output,int number_of_channels, int image_height, int image_width,int blockwidth )
{

	__shared__ float Ns[32][32];


	for( int curr_channel=0; curr_channel<number_of_channels; curr_channel++)
	{

    Ns[threadIdx.x][threadIdx.y] = image[(threadIdx.y*number_of_channels +curr_channel +blockIdx.y * (blockwidth*number_of_channels)) + (threadIdx.x + blockIdx.x*blockwidth)* (image_width *number_of_channels) ];

    __syncthreads();

    if((threadIdx.x % 2 == 0) && (threadIdx.y %2 == 0))
    {
      output[blockIdx.y*(blockwidth/2) *number_of_channels+ (threadIdx.y/2) *number_of_channels+ curr_channel + (blockIdx.x * blockwidth/2 +threadIdx.x/2) * (image_width/2)*number_of_channels] = max4(Ns[threadIdx.x][threadIdx.y],Ns[threadIdx.x][threadIdx.y+1],Ns[threadIdx.x+1][threadIdx.y],Ns[threadIdx.x+1][threadIdx.y+1]);
    }
  }
}
//@@ INSERT CODE HERE
__global__ void fully1(float *I, const float* __restrict__ M, float *P,int channels,int outputChannels,float *b)
{
   __shared__ float F_ds[7][7];

   float acc[4096] ={0};

     for (int current_channel = 0; current_channel < channels; current_channel++)
     {

         F_ds[threadIdx.x][threadIdx.y] = I[(threadIdx.x + (blockIdx.x * TILE_WIDTH)) * (7 * channels) + threadIdx.y * channels + current_channel + blockIdx.y * (TILE_WIDTH) ]; //
        __syncthreads();

        int y, x,z;
        if(threadIdx.y == 0 && threadIdx.x ==0)
        {

          for(int z =0;z<outputChannels;z++)
          {
              for(int i =0; i< 7; i++)
              {
                for(int j =0; j<7; j++)
                {
                   acc[z] += F_ds[i][j] * M[z*7*7*512 + current_channel*7*7 +i*7+j];
                }
            }

          }
       }

   }
   if(threadIdx.y == 0 && threadIdx.x ==0)
   {
     for(int z =0;z<outputChannels;z++)
     {

        P[z] = acc[z] +b[z];
     }

   }
}

__global__ void fully2(float *I, const float* __restrict__ M, float *P,int channels,int outputChannels,float *b)
{
   __shared__ float F_ds[4][32][32];

   float acc[4096] ={0};

    for(int i=0;i<4;i++)
    {
      F_ds[i][threadIdx.x][threadIdx.y]=I[threadIdx.y + threadIdx.x*32 +i*32*32];
    }

     __syncthreads();

     if(threadIdx.y == 0 && threadIdx.x ==0)
     {
        int i,j,k;

          for (int current_op_channel = 0; current_op_channel < outputChannels; current_op_channel++)
          {

                 for(int current_channel =0; current_channel<channels;current_channel++)
                 {
                      i=current_channel/1024;
                      j=((current_channel)/32)%32;
                      k=current_channel%32;
                      acc[current_op_channel]+=F_ds[i][j][k]*M[current_channel +current_op_channel*channels];
                 }
           }

         for(int z =0;z<outputChannels;z++)
         {
              P[z] = clamp(acc[z] + b[z]);
         }

      }
}

__global__ void fully3(float *I, const float* __restrict__ M, float *P,int channels,int outputChannels,float *b)
{
   __shared__ float F_ds[4][32][32];

   float acc[1000] ={0};

    for(int i=0;i<4;i++)
    {
      F_ds[i][threadIdx.x][threadIdx.y]=I[threadIdx.y + threadIdx.x*32 +i*32*32];
    }

     __syncthreads();

     if(threadIdx.y == 0 && threadIdx.x ==0)
     {
        int i,j,k;
         for (int current_op_channel = 0; current_op_channel < outputChannels; current_op_channel++)
         {

                for(int current_channel =0; current_channel<channels;current_channel++)
                {
                  i=current_channel/1024;
                  j=((current_channel)/32)%32;
                  k=current_channel%32;

                  acc[current_op_channel]+=F_ds[i][j][k]*M[current_channel +current_op_channel*channels];

                }
          }

         for(int z =0;z<outputChannels;z++)
         {
                       P[z] = clamp(acc[z] + b[z]);
         }

      }
}

// in first go, all of the threads will load the image pixels TILE_WIDTH * TILE_WIDTH on the second go first (TILE_WIDTH-mask radius)^2 threads will load the image.
__global__ void convolution(float *I, const float* __restrict__ M, float *P, float *b,int channels, int width, int height,int outputChannels)
{
   __shared__ float N_ds[B_y][B_x];
   int k;int dest_Y;int dest_X;int src_X; int src_Y;int src;

   float accum[out] = {0};

   // for all the image channels
   for (current_channel = 0; current_channel < channels; current_channel++)
   {


       dest = threadIdx.y * TILE_WIDTH + threadIdx.x,
      // The new index of thread in matrix with the boundary
      dest_Y = dest / B_x,
      dest_X = dest % B_x,

      src_Y = blockIdx.y * TILE_WIDTH + dest_Y - Mask_radius_x,
      src_X = blockIdx.x * TILE_WIDTH + dest_X - Mask_radius_y,
      src = (src_Y * width + src_X) * channels + current_channel;
      if (src_Y >= 0 && src_Y < height && src_X >= 0 && src_X < width)
         N_ds[dest_Y][dest_X] = I[src];
      else
         N_ds[dest_Y][dest_X] = 0.0;

        for (int iter=1; iter <= (B_x * B_y) / (TILE_WIDTH*TILE_WIDTH); iter++)
        {
           // Second batch loading
           dest = threadIdx.y * TILE_WIDTH + threadIdx.x + iter*(TILE_WIDTH * TILE_WIDTH);
            dest_Y = dest / B_x, dest_X = dest % B_x;
            src_Y  = blockIdx.y * TILE_WIDTH + dest_Y - Mask_radius_x;
            src_X = blockIdx.x * TILE_WIDTH + dest_X - Mask_radius_y;
            src = (src_Y * width + src_X) * channels + current_channel;
            if (dest_Y < B_y && dest_X < B_x)
            {
                if (src_Y >= 0 && src_Y < height && src_X >= 0 && src_X < width)
                    N_ds[dest_Y][dest_X] = I[src];
                else
                    N_ds[dest_Y][dest_X] = 0.0;
            }
        }
      __syncthreads();

      int y, x,z;
      for(z =0;z<outputChannels;z++)
        for (y = 0; y < Mask_width; y++)
           for (x = 0; x < Mask_width; x++)
              //                                                                                        navigation with input channel mask  inside mask navigate
              accum[z] += N_ds[threadIdx.y + y][threadIdx.x + x] * M[ ( z*Mask_width*Mask_width*channels + current_channel*Mask_width*Mask_width) + y * Mask_width + x];

      __syncthreads();
   }

   int y, x,z;
   y = blockIdx.y * TILE_WIDTH + threadIdx.y;
   x = blockIdx.x * TILE_WIDTH + threadIdx.x;
   if (y < height && x < width)
   // add bias and relu
      for(z =0;z<outputChannels;z++)
          P[(y * width*outputChannels + outputChannels*x)+z] = clamp(accum[z]  + b[z]);
}

float convolution_2D_OnHost(float * N,float * M,int width, int height,int i,int j,int imageChannels ,int outputChannels);

int main()
{
     // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;


    err = hipDeviceReset();
    if(err != hipSuccess)
    {
      printf("failed to reset device \n");
    }

    float time_taken = 0.0f;
     hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int maskRows=Mask_height; // Set it as per requirement of 64 X 32
    int maskColumns=Mask_width;

    int imageChannels=3;
    int outputChannels = 64;
    int imageWidth=SIZE;
    int imageHeight=SIZE;

    float * hostOutputImageData;
    float * hostOutputMaxPooledData;
    float * deviceInputImageData;
    float * deviceOutputImageData_1_1;
    float * deviceOutputMaxPooledData;
  //  float * deviceInputImageData;
    float * deviceMaskData;
    //float * outputImageOnHost;
    float * deviceInputMaxPool;
	float * bias;
	float * deviceBias;
  float *biasDense = (float *) malloc(sizeof(float)*dense[0][1]);

    bias = (float *) malloc(sizeof(float)*layers[12][0]);
    /*************************** conv1-1 ******************************/
    int level = 0;
    // layer parameters
    outputChannels = layers[level][0];
    imageChannels = layers[level][1];


    float * hostMaskData = (float *) malloc(sizeof(float)*outputChannels*imageChannels*CONV_SIZE*CONV_SIZE);
    readWeights(level,hostMaskData, bias);


    //To store Memory

    hostOutputImageData = (float *) malloc(sizeof(float)*imageWidth*imageHeight*outputChannels);
    //outputImageOnHost = (float *) malloc(sizeof(float)*imageWidth*imageHeight*outputChannels);

    float * hostInputImageData = (float*) malloc (sizeof (float) * SIZE * SIZE * INPUT_CHANNELS);
    normalizeBGR (hostInputImageData);

    //wbCheck(hipMalloc((void **) &deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float)));
	err = hipMalloc((void**)&deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate deviceInputImageData (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **) &deviceOutputImageData_1_1, imageWidth * imageHeight *outputChannels* sizeof(float));
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate deviceOutputImageData(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * imageChannels*outputChannels* sizeof(float));
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate deviceMaskData(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	err = hipMalloc((void**)&deviceBias, imageWidth * imageHeight * imageChannels * sizeof(float));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate deviceBias (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	printf("Copy input data from the host memory to the CUDA device level 1_1\n");
    err = hipMemcpy(deviceInputImageData, hostInputImageData,
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy input matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(deviceMaskData,
               hostMaskData,
               outputChannels*imageChannels*CONV_SIZE*CONV_SIZE*sizeof(float),
               hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy mask matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	err = hipMemcpy(deviceBias, bias,
                sizeof(float)*layers[12][0],
               hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy input matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    dim3 dimGrid(((imageWidth-1)/TILE_WIDTH)+1, ((imageHeight-1)/TILE_WIDTH)+1,1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

    hipEventRecord(start);

    convolution<<<dimGrid,dimBlock>>>(deviceInputImageData, deviceMaskData, deviceOutputImageData_1_1, deviceBias,
                                       imageChannels, imageWidth, imageHeight,outputChannels);


    hipMemcpy(hostOutputImageData,
              deviceOutputImageData_1_1,
              imageWidth * imageHeight * outputChannels * sizeof(float),
              hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    FILE *level0;
    if ((level0 = fopen("level0.txt","w")) == NULL){
        printf("Error! opening device file");
    exit(1);
    }
    for(int i=0;i<outputChannels*imageChannels*CONV_SIZE*CONV_SIZE;i++)
    {
        if(i>0 && (i%imageWidth==0))
        {
             fprintf(level0,"\n");

        }
      fprintf(level0, "%0.2f \t", *(hostMaskData+i));


    }
    fclose(level0);

   // Free conv_1_1 Memory
   free(hostMaskData);
   hipFree(deviceMaskData);
   hipFree(deviceBias);
   free(hostOutputImageData);
   hipFree(deviceInputImageData);
   free(hostInputImageData);
     /*************************** conv1-1 end******************************/
     /*************************** conv1-2 start ******************************/
     level = 1;
     // layer parameters

     outputChannels = layers[level][0];
     imageChannels = layers[level][1];

     float * deviceOutputImageData_1_2;

     hostMaskData = (float *) malloc(sizeof(float)*outputChannels*imageChannels*CONV_SIZE*CONV_SIZE);
     hostOutputImageData = (float *) malloc(sizeof(float)*imageWidth*imageHeight*outputChannels);

     err = hipMalloc((void**)&deviceBias, imageWidth * imageHeight * imageChannels * sizeof(float));
      if (err != hipSuccess)
      {
          fprintf(stderr, "Failed to allocate deviceBias (error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
      }

     err = hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * imageChannels*outputChannels* sizeof(float));
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to allocate deviceMaskData(error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }

    err = hipMalloc((void **) &deviceOutputImageData_1_2, imageWidth * imageHeight *outputChannels* sizeof(float));
    if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to allocate deviceOutputImageData(error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
     readWeights(level,hostMaskData, bias);

     printf("Copy input data from the host memory to the CUDA device level 1_2\n");
     // Copy device bias
     err = hipMemcpy(deviceBias, bias,
                   sizeof(float)*layers[12][0],
                  hipMemcpyHostToDevice);
       if (err != hipSuccess)
       {
           fprintf(stderr, "Failed to copy input matrix from host to device (error code %s)!\n", hipGetErrorString(err));
           exit(EXIT_FAILURE);
       }
      // Copy device mask
       err = hipMemcpy(deviceMaskData,
                  hostMaskData,
                  outputChannels*imageChannels*CONV_SIZE*CONV_SIZE*sizeof(float),
                  hipMemcpyHostToDevice);

       if (err != hipSuccess)
       {
           fprintf(stderr, "Failed to copy mask matrix from host to device (error code %s)!\n", hipGetErrorString(err));
           exit(EXIT_FAILURE);
       }

     convolution<<<dimGrid,dimBlock>>>(deviceOutputImageData_1_1, deviceMaskData, deviceOutputImageData_1_2, deviceBias,
                                        imageChannels, imageWidth, imageHeight,outputChannels);
     hipMemcpy(hostOutputImageData,
              deviceOutputImageData_1_2,
              imageWidth * imageHeight * outputChannels * sizeof(float),
              hipMemcpyDeviceToHost);
     hipDeviceSynchronize();
      // Program exits if the file pointer returns NULL.
      FILE *out;
      if ((out = fopen("device_conv.txt","w")) == NULL){
          printf("Error! opening device file");
      exit(1);
      }
      for(int i=0;i<imageWidth*imageHeight*outputChannels;i++)
      {
      	  if(i>0 && (i%imageWidth==0))
      	  {
      	       fprintf(out,"\n");

      	  }
	      fprintf(out, "%0.2f \t", *(hostOutputImageData+i));


      }
      fclose(out);

      FILE *level1;
      if ((level1 = fopen("level1.txt","w")) == NULL){
          printf("Error! opening device file");
      exit(1);
      }
      for(int i=0;i<outputChannels*imageChannels*CONV_SIZE*CONV_SIZE;i++)
      {
          if(i>0 && (i%imageWidth==0))
          {
               fprintf(level1,"\n");

          }
        fprintf(level1, "%0.2f \t", *(hostMaskData+i));


      }
      fclose(level1);

      // Free conv_1_1 Memory
      free(hostMaskData);
      hipFree(deviceMaskData);
      hipFree(deviceBias);
      free(hostOutputImageData);
      hipFree(deviceOutputImageData_1_1);

     /*************************** conv1-2 end ******************************/
     /*************************** conv1-maxpool start ******************************/
     // Layer parameters

     hostOutputMaxPooledData = (float *) malloc(sizeof(float)*imageWidth/2*imageHeight/2*outputChannels);
     err = hipMalloc((void**) &deviceOutputMaxPooledData, imageWidth/2 * imageHeight/2 * outputChannels * sizeof(float));
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to allocate deviceInputImageData (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
    err = hipMalloc((void**) &deviceInputMaxPool, imageWidth * imageHeight * outputChannels * sizeof(float));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate deviceInputImageData (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(deviceInputMaxPool, hostOutputImageData,
               imageWidth * imageHeight * outputChannels * sizeof(float),
               hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy input matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // image 224
    int blockwidth = 32;
    int number_blocks = imageWidth/blockwidth;
    dim3 dimGrid_m1(number_blocks,number_blocks,1);
    dim3 dimBlock_m1(blockwidth,blockwidth,1);
  maxpool<<<dimGrid_m1,dimBlock_m1>>>(deviceOutputImageData_1_2,deviceOutputMaxPooledData ,outputChannels, imageHeight, imageWidth,blockwidth);


  hipDeviceSynchronize();

  hipMemcpy(hostOutputMaxPooledData,
             deviceOutputMaxPooledData,
             imageWidth/2 * imageHeight/2 * outputChannels * sizeof(float),
             hipMemcpyDeviceToHost);

     FILE *mp;

     if ((mp = fopen("maxpooled.txt","w")) == NULL){
         printf("Error! opening host file");

         // Program exits if the file pointer returns NULL.
         exit(1);
     }

      for(int i=0;i<imageWidth/2*imageHeight/2*outputChannels;i++)
         {
              if(i>0 && (i%(imageWidth/2 * outputChannels)==0))
                 fprintf(mp,"\n");

           fprintf(mp, "%0.2f \t", *(hostOutputMaxPooledData+i));
         }

  //  hipFree(deviceOutputMaxPooledData);
    hipFree(deviceOutputImageData_1_2);
    hipFree(deviceInputMaxPool);

    free(hostOutputMaxPooledData);
/********************************conv_1 max end**********************************************/

/*******************************conv_2_1 start**********************************************/
// Layer parameters
  imageWidth /= 2;
  imageHeight /= 2;

  // Layer 4 (Convolution 64 -> 128)
  level = 2;
  outputChannels = layers[level][0];
  imageChannels = layers[level][1];


  float * deviceOutputImageData_2_1;

  hostMaskData = (float *) malloc(sizeof(float)*outputChannels*imageChannels*CONV_SIZE*CONV_SIZE);
  hostOutputImageData = (float *) malloc(sizeof(float)*imageWidth*imageHeight*outputChannels);
  err = hipMalloc((void**)&deviceBias, imageWidth * imageHeight * imageChannels * sizeof(float));
   if (err != hipSuccess)
   {
       fprintf(stderr, "Failed to allocate deviceBias (error code %s)!\n", hipGetErrorString(err));
       exit(EXIT_FAILURE);
   }


  err = hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * imageChannels*outputChannels* sizeof(float));
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate deviceMaskData(error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

 err = hipMalloc((void **) &deviceOutputImageData_2_1, imageWidth * imageHeight *outputChannels* sizeof(float));
 if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate deviceOutputImageData(error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
  readWeights(level,hostMaskData, bias);

  printf("Copy input data from the host memory to the CUDA device level 2_1\n");
  // Copy device bias
  err = hipMemcpy(deviceBias, bias,
                sizeof(float)*layers[12][0],
               hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy input matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
   // Copy device mask
    err = hipMemcpy(deviceMaskData,
               hostMaskData,
               outputChannels*imageChannels*CONV_SIZE*CONV_SIZE*sizeof(float),
               hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy mask matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    dim3 dimGrid_2(((imageWidth-1)/TILE_WIDTH)+1, ((imageHeight-1)/TILE_WIDTH)+1,1);
    dim3 dimBlock_2(TILE_WIDTH, TILE_WIDTH, 1);
  convolution<<<dimGrid_2,dimBlock_2>>>(deviceOutputMaxPooledData, deviceMaskData, deviceOutputImageData_2_1, deviceBias,
                                     imageChannels, imageWidth, imageHeight,outputChannels);
  hipMemcpy(hostOutputImageData,
           deviceOutputImageData_2_1,
           imageWidth * imageHeight * outputChannels * sizeof(float),
           hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
   // Program exits if the file pointer returns NULL.
   FILE *out2_1;
   if ((out2_1 = fopen("out2_1.txt","w")) == NULL){
       printf("Error! opening device file");
   exit(1);
   }
   for(int i=0;i<imageWidth*imageHeight*outputChannels;i++)
   {
       if(i>0 && (i%imageWidth==0))
       {
            fprintf(out2_1,"\n");

       }
     fprintf(out2_1, "%0.2f \t", *(hostOutputImageData+i));


   }
   fclose(out2_1);

   FILE *level2_1;
   if ((level2_1 = fopen("level2_1.txt","w")) == NULL){
       printf("Error! opening device file");
   exit(1);
   }
   for(int i=0;i<outputChannels*imageChannels*CONV_SIZE*CONV_SIZE;i++)
   {
       if(i>0 && (i%imageWidth==0))
       {
            fprintf(level2_1,"\n");

       }
     fprintf(level2_1, "%0.5f \t", *(hostMaskData+i));


   }
   fclose(level2_1);

   // Free conv_2_1 Memory
  free(hostMaskData);
   hipFree(deviceMaskData);
   free(hostOutputImageData);
   hipFree(deviceBias);
   //hipFree(deviceOutputMaxPooledData);


/*******************************conv_2_1 end**********************************************/

/******************************conv_2_2 start********************************************/

  // Layer 4 (Convolution 128-> 128)
  level = 3;
  outputChannels = layers[level][0];
  imageChannels = layers[level][1];


  float * deviceOutputImageData_2_2;

  hostMaskData = (float *) malloc(sizeof(float)*outputChannels*imageChannels*CONV_SIZE*CONV_SIZE);
  hostOutputImageData = (float *) malloc(sizeof(float)*imageWidth*imageHeight*outputChannels);
  err = hipMalloc((void**)&deviceBias, imageWidth * imageHeight * imageChannels * sizeof(float));
   if (err != hipSuccess)
   {
       fprintf(stderr, "Failed to allocate deviceBias (error code %s)!\n", hipGetErrorString(err));
       exit(EXIT_FAILURE);
   }


  err = hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * imageChannels*outputChannels* sizeof(float));
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate deviceMaskData(error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

 err = hipMalloc((void **) &deviceOutputImageData_2_2, imageWidth * imageHeight *outputChannels* sizeof(float));
 if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate deviceOutputImageData(error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
  readWeights(level,hostMaskData, bias);

  printf("Copy input data from the host memory to the CUDA device level 2_1\n");
  // Copy device bias
  err = hipMemcpy(deviceBias, bias,
                sizeof(float)*layers[12][0],
               hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy input matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
   // Copy device mask
    err = hipMemcpy(deviceMaskData,
               hostMaskData,
               outputChannels*imageChannels*CONV_SIZE*CONV_SIZE*sizeof(float),
               hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy mask matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

  convolution<<<dimGrid_2,dimBlock_2>>>(deviceOutputImageData_2_1, deviceMaskData, deviceOutputImageData_2_2, deviceBias,
                                     imageChannels, imageWidth, imageHeight,outputChannels);
  hipMemcpy(hostOutputImageData,
           deviceOutputImageData_2_2,
           imageWidth * imageHeight * outputChannels * sizeof(float),
           hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
   // Program exits if the file pointer returns NULL.
   FILE *out2_2;
   if ((out2_2 = fopen("out2_2.txt","w")) == NULL){
       printf("Error! opening device file");
   exit(1);
   }
   for(int i=0;i<imageWidth*imageHeight*outputChannels;i++)
   {
       if(i>0 && (i%imageWidth==0))
       {
            fprintf(out2_2,"\n");

       }
     fprintf(out2_2, "%0.2f \t", *(hostOutputImageData+i));


   }
   fclose(out2_2);

   FILE *level2_2;
   if ((level2_2 = fopen("level2_2.txt","w")) == NULL){
       printf("Error! opening device file");
   exit(1);
   }
   for(int i=0;i<outputChannels*imageChannels*CONV_SIZE*CONV_SIZE;i++)
   {
       if(i>0 && (i%imageWidth==0))
       {
            fprintf(level2_2,"\n");

       }
     fprintf(level2_2, "%0.5f \t", *(hostMaskData+i));


   }
   fclose(level2_2);

   // Free conv_2_1 Memory
  free(hostMaskData);
   hipFree(deviceMaskData);
   hipFree(deviceBias);
   free(hostOutputImageData);
   hipFree(deviceOutputImageData_2_1);

/******************************conv_2_2 end*********************************************/

/******************************max2 start**********************************************/
    float * deviceOutputMaxPooledData2;
     hostOutputMaxPooledData = (float *) malloc(sizeof(float)*imageWidth/2*imageHeight/2*outputChannels);
     err = hipMalloc((void**) &deviceOutputMaxPooledData2, imageWidth/2 * imageHeight/2 * outputChannels * sizeof(float));
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to allocate deviceInputImageData (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
     // image 112
     blockwidth = 16;
     number_blocks = imageWidth/blockwidth;
     dim3 dimGrid_m2(number_blocks,number_blocks,1);
     dim3 dimBlock_m2(blockwidth,blockwidth,1);

  maxpool<<<dimGrid_m2,dimBlock_m2>>>(deviceOutputImageData_2_2,deviceOutputMaxPooledData2 ,outputChannels, imageHeight, imageWidth,blockwidth);


  hipDeviceSynchronize();

  hipMemcpy(hostOutputMaxPooledData,
             deviceOutputMaxPooledData2,
             imageWidth/2 * imageHeight/2 * outputChannels * sizeof(float),
             hipMemcpyDeviceToHost);

     FILE *mp_2;

     if ((mp_2 = fopen("maxpooled2.txt","w")) == NULL){
         printf("Error! opening host file");

         // Program exits if the file pointer returns NULL.
         exit(1);
     }

      for(int i=0;i<imageWidth/2*imageHeight/2*outputChannels;i++)
         {
              if(i>0 && (i%(imageWidth/2 * outputChannels)==0))
                 fprintf(mp_2,"\n");

           fprintf(mp_2, "%0.2f \t", *(hostOutputMaxPooledData+i));
         }

  //  hipFree(deviceOutputMaxPooledData);
    hipFree(deviceOutputImageData_2_2);

    //hipFree(deviceInputMaxPool);
    free(hostOutputMaxPooledData);
/*****************************max2 end*************************************************/

/*****************************conv_3_1 start************************************************/
// Layer parameters
  imageWidth /= 2;
  imageHeight /= 2;

  // Layer 4 (Convolution 128 -> 256)
  level = 4;
  outputChannels = layers[level][0];
  imageChannels = layers[level][1];


  float * deviceOutputImageData_3_1;

  hostMaskData = (float *) malloc(sizeof(float)*outputChannels*imageChannels*CONV_SIZE*CONV_SIZE);
  float *hostOutputImageData1 = (float *) malloc(sizeof(float)*imageWidth*imageHeight*outputChannels);

  err = hipMalloc((void**)&deviceBias, imageWidth * imageHeight * imageChannels * sizeof(float));
   if (err != hipSuccess)
   {
       fprintf(stderr, "Failed to allocate deviceBias (error code %s)!\n", hipGetErrorString(err));
       exit(EXIT_FAILURE);
   }


  err = hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * imageChannels*outputChannels* sizeof(float));
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate deviceMaskData(error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

 err = hipMalloc((void **) &deviceOutputImageData_3_1, imageWidth * imageHeight *outputChannels* sizeof(float));
 if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate deviceOutputImageData(error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
  readWeights(level,hostMaskData, bias);

  printf("Copy input data from the host memory to the CUDA device level 3_1\n");

  // Copy device bias
  err = hipMemcpy(deviceBias, bias,
                sizeof(float)*layers[12][0],
               hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy input matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
   // Copy device mask
    err = hipMemcpy(deviceMaskData,
               hostMaskData,
               outputChannels*imageChannels*CONV_SIZE*CONV_SIZE*sizeof(float),
               hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy mask matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    dim3 dimGrid_3(((imageWidth-1)/TILE_WIDTH)+1, ((imageHeight-1)/TILE_WIDTH)+1,1);
    dim3 dimBlock_3(TILE_WIDTH, TILE_WIDTH, 1);
    convolution<<<dimGrid_3,dimBlock_3>>>(deviceOutputMaxPooledData2, deviceMaskData, deviceOutputImageData_3_1, deviceBias,
                                     imageChannels, imageWidth, imageHeight,outputChannels);
   err =   hipMemcpy(hostOutputImageData1,
           deviceOutputImageData_3_1,
           imageWidth * imageHeight * outputChannels * sizeof(float),
           hipMemcpyDeviceToHost);

   if (err != hipSuccess)
   {
       fprintf(stderr, "Failed to copy 3.1 (error code %s)!\n", hipGetErrorString(err));
       exit(EXIT_FAILURE);
   }
  hipDeviceSynchronize();
   // Program exits if the file pointer returns NULL.
   FILE *out3_1;
   if ((out3_1 = fopen("out3_1.txt","w")) == NULL){
       printf("Error! opening device file");
   exit(1);
   }
   for(int i=0;i<imageWidth*imageHeight*outputChannels;i++)
   {
       if(i>0 && (i%imageWidth==0))
       {
            fprintf(out3_1,"\n");

       }
     fprintf(out3_1, "%0.2f \t", *(hostOutputImageData1+i));


   }
   fclose(out3_1);

   FILE *level3_1;
   if ((level3_1 = fopen("level3_1.txt","w")) == NULL){
       printf("Error! opening device file");
   exit(1);
   }
   for(int i=0;i<outputChannels*imageChannels*CONV_SIZE*CONV_SIZE;i++)
   {
       if(i>0 && (i%imageWidth==0))
       {
            fprintf(level3_1,"\n");

       }
     fprintf(level3_1, "%0.5f \t", *(hostMaskData+i));


   }
   fclose(level3_1);

   // Free conv_2_1 Memory
  //free(hostMaskData);
   hipFree(deviceMaskData);
   free(hostOutputImageData1);
   hipFree(deviceOutputMaxPooledData2);
   hipFree(deviceBias);
// /*****************************conv_3_1 end************************************************/
//
// /****************************conv_3_2 start**********************************************/
// Layer 4 (Convolution 128-> 128)
level = 5;
outputChannels = layers[level][0];
imageChannels = layers[level][1];

hostOutputImageData = (float *) malloc(sizeof(float)*imageWidth*imageHeight*outputChannels);
float * deviceOutputImageData_3_2;

hostMaskData = (float *) malloc(sizeof(float)*outputChannels*imageChannels*CONV_SIZE*CONV_SIZE);
//hostOutputImageData = (float *) malloc(sizeof(float)*imageWidth*imageHeight*outputChannels);
err = hipMalloc((void**)&deviceBias, imageWidth * imageHeight * imageChannels * sizeof(float));
 if (err != hipSuccess)
 {
     fprintf(stderr, "Failed to allocate deviceBias (error code %s)!\n", hipGetErrorString(err));
     exit(EXIT_FAILURE);
 }


err = hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * imageChannels*outputChannels* sizeof(float));
if (err != hipSuccess)
{
    fprintf(stderr, "Failed to allocate deviceMaskData(error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
}

err = hipMalloc((void **) &deviceOutputImageData_3_2, imageWidth * imageHeight *outputChannels* sizeof(float));
if (err != hipSuccess)
{
    fprintf(stderr, "Failed to allocate deviceOutputImageData(error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
}
readWeights(level,hostMaskData, bias);

printf("Copy input data from the host memory to the CUDA device level 3_2\n");
// Copy device bias
err = hipMemcpy(deviceBias, bias,
              sizeof(float)*layers[12][0],
             hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy input matrix from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
 // Copy device mask
  err = hipMemcpy(deviceMaskData,
             hostMaskData,
             outputChannels*imageChannels*CONV_SIZE*CONV_SIZE*sizeof(float),
             hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy mask matrix from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

convolution<<<dimGrid_3,dimBlock_3>>>(deviceOutputImageData_3_1, deviceMaskData, deviceOutputImageData_3_2, deviceBias,
                                   imageChannels, imageWidth, imageHeight,outputChannels);
hipMemcpy(hostOutputImageData,
         deviceOutputImageData_3_2,
         imageWidth * imageHeight * outputChannels * sizeof(float),
         hipMemcpyDeviceToHost);
hipDeviceSynchronize();
 // Program exits if the file pointer returns NULL.
 FILE *out3_2;
 if ((out3_2 = fopen("out3_2.txt","w")) == NULL){
     printf("Error! opening device file");
 exit(1);
 }
 for(int i=0;i<imageWidth*imageHeight*outputChannels;i++)
 {
     if(i>0 && (i%imageWidth==0))
     {
          fprintf(out3_2,"\n");

     }
   fprintf(out3_2, "%0.2f \t", *(hostOutputImageData+i));


 }
 fclose(out3_2);

 // Free conv_2_1 Memory
free(hostMaskData);
 hipFree(deviceMaskData);
 hipFree(deviceBias);
// free(hostOutputImageData);
 hipFree(deviceOutputImageData_3_1);
/***************************conv_3_2 end************************************************/
/***************************conv_3_3 start************************************************/
// Layer 4 (Convolution 128-> 128)
level = 6;
outputChannels = layers[level][0];
imageChannels = layers[level][1];


float * deviceOutputImageData_3_3;

hostMaskData = (float *) malloc(sizeof(float)*outputChannels*imageChannels*CONV_SIZE*CONV_SIZE);
hostOutputImageData = (float *) malloc(sizeof(float)*imageWidth*imageHeight*outputChannels);

err = hipMalloc((void**)&deviceBias, imageWidth * imageHeight * imageChannels * sizeof(float));
 if (err != hipSuccess)
 {
     fprintf(stderr, "Failed to allocate deviceBias (error code %s)!\n", hipGetErrorString(err));
     exit(EXIT_FAILURE);
 }


err = hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * imageChannels*outputChannels* sizeof(float));
if (err != hipSuccess)
{
    fprintf(stderr, "Failed to allocate deviceMaskData(error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
}

err = hipMalloc((void **) &deviceOutputImageData_3_3, imageWidth * imageHeight *outputChannels* sizeof(float));
if (err != hipSuccess)
{
    fprintf(stderr, "Failed to allocate deviceOutputImageData(error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
}
readWeights(level,hostMaskData, bias);

printf("Copy input data from the host memory to the CUDA device level 3_3\n");
// Copy device bias
err = hipMemcpy(deviceBias, bias,
              sizeof(float)*layers[12][0],
             hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy input matrix from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
 // Copy device mask
  err = hipMemcpy(deviceMaskData,
             hostMaskData,
             outputChannels*imageChannels*CONV_SIZE*CONV_SIZE*sizeof(float),
             hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy mask matrix from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

convolution<<<dimGrid_3,dimBlock_3>>>(deviceOutputImageData_3_2, deviceMaskData, deviceOutputImageData_3_3, deviceBias,
                                   imageChannels, imageWidth, imageHeight,outputChannels);
hipMemcpy(hostOutputImageData,
         deviceOutputImageData_3_3,
         imageWidth * imageHeight * outputChannels * sizeof(float),
         hipMemcpyDeviceToHost);
hipDeviceSynchronize();
 // Program exits if the file pointer returns NULL.
 FILE *out3_3;
 if ((out3_3 = fopen("out3_2.txt","w")) == NULL){
     printf("Error! opening device file");
 exit(1);
 }
 for(int i=0;i<imageWidth*imageHeight*outputChannels;i++)
 {
     if(i>0 && (i%imageWidth==0))
     {
          fprintf(out3_3,"\n");

     }
   fprintf(out3_3, "%0.2f \t", *(hostOutputImageData+i));


 }
 fclose(out3_3);

 // Free conv_2_1 Memory
free(hostMaskData);
 hipFree(deviceMaskData);
 hipFree(deviceBias);
free(hostOutputImageData);
 hipFree(deviceOutputImageData_3_2);

/***************************conv_3_3 end************************************************/

/******************************max3 start**********************************************/
    float * deviceOutputMaxPooledData3;
     hostOutputMaxPooledData = (float *) malloc(sizeof(float)*imageWidth/2*imageHeight/2*outputChannels);
     err = hipMalloc((void**) &deviceOutputMaxPooledData3, imageWidth/2 * imageHeight/2 * outputChannels * sizeof(float));
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to allocate deviceInputImageData (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
     // image 56
     blockwidth = 8;
     number_blocks = imageWidth/blockwidth;
     dim3 dimGrid_m3(number_blocks,number_blocks,1);
     dim3 dimBlock_m3(blockwidth,blockwidth,1);

  maxpool<<<dimGrid_m3,dimBlock_m3>>>(deviceOutputImageData_3_3,deviceOutputMaxPooledData3 ,outputChannels, imageHeight, imageWidth, blockwidth);


  hipDeviceSynchronize();

  hipMemcpy(hostOutputMaxPooledData,
             deviceOutputMaxPooledData3,
             imageWidth/2 * imageHeight/2 * outputChannels * sizeof(float),
             hipMemcpyDeviceToHost);

     FILE *mp_3;

     if ((mp_3 = fopen("maxpooled2.txt","w")) == NULL){
         printf("Error! opening host file");

         // Program exits if the file pointer returns NULL.
         exit(1);
     }

      for(int i=0;i<imageWidth/2*imageHeight/2*outputChannels;i++)
         {
              if(i>0 && (i%(imageWidth/2 * outputChannels)==0))
                 fprintf(mp_3,"\n");

           fprintf(mp_3, "%0.2f \t", *(hostOutputMaxPooledData+i));
         }

  //  hipFree(deviceOutputMaxPooledData);
    hipFree(deviceOutputImageData_3_3);

  //  hipFree(deviceInputMaxPool);
    free(hostOutputMaxPooledData);
/*****************************max3 end*************************************************/

/*****************************conv_4_1 start************************************************/
// Layer parameters
 imageWidth /= 2;
  imageHeight /= 2;

  // Layer 4 (Convolution 128 -> 256)
  level = 7;
  outputChannels = layers[level][0];
  imageChannels = layers[level][1];


  float * deviceOutputImageData_4_1;

  hostMaskData = (float *) malloc(sizeof(float)*outputChannels*imageChannels*CONV_SIZE*CONV_SIZE);
  hostOutputImageData = (float *) malloc(sizeof(float)*imageWidth*imageHeight*outputChannels);
  err = hipMalloc((void**)&deviceBias, imageWidth * imageHeight * imageChannels * sizeof(float));
   if (err != hipSuccess)
   {
       fprintf(stderr, "Failed to allocate deviceBias (error code %s)!\n", hipGetErrorString(err));
       exit(EXIT_FAILURE);
   }


  err = hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * imageChannels*outputChannels* sizeof(float));
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate deviceMaskData(error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

 err = hipMalloc((void **) &deviceOutputImageData_4_1, imageWidth * imageHeight *outputChannels* sizeof(float));
 if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate deviceOutputImageData(error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
  readWeights(level,hostMaskData, bias);

  printf("Copy input data from the host memory to the CUDA device level 4_1\n");
  // Copy device bias
  err = hipMemcpy(deviceBias, bias,
                sizeof(float)*layers[12][0],
               hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy input matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
   // Copy device mask
    err = hipMemcpy(deviceMaskData,
               hostMaskData,
               outputChannels*imageChannels*CONV_SIZE*CONV_SIZE*sizeof(float),
               hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy mask matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    dim3 dimGrid_4(((imageWidth-1)/TILE_WIDTH)+1, ((imageHeight-1)/TILE_WIDTH)+1,1);
    dim3 dimBlock_4(TILE_WIDTH, TILE_WIDTH, 1);
    convolution<<<dimGrid_4,dimBlock_4>>>(deviceOutputMaxPooledData3, deviceMaskData, deviceOutputImageData_4_1, deviceBias,
                                     imageChannels, imageWidth, imageHeight,outputChannels);
   err =   hipMemcpy(hostOutputImageData,
           deviceOutputImageData_4_1,
           imageWidth * imageHeight * outputChannels * sizeof(float),
           hipMemcpyDeviceToHost);

  if (err != hipSuccess)
   {
       fprintf(stderr, "Failed to copy 4.1 (error code %s)!\n", hipGetErrorString(err));
       exit(EXIT_FAILURE);
   }
  hipDeviceSynchronize();
   // Program exits if the file pointer returns NULL.
   FILE *out4_1;
   if ((out4_1 = fopen("out4_1.txt","w")) == NULL){
       printf("Error! opening device file");
   exit(1);
   }
   for(int i=0;i<imageWidth*imageHeight*outputChannels;i++)
   {
       if(i>0 && (i%imageWidth==0))
       {
            fprintf(out4_1,"\n");

       }
     fprintf(out4_1, "%0.2f \t", *(hostOutputImageData1+i));


   }
   fclose(out4_1);

   FILE *level4_1;
   if ((level4_1 = fopen("level4_1.txt","w")) == NULL){
       printf("Error! opening device file");
   exit(1);
 }

   for(int i=0;i<outputChannels*imageChannels*CONV_SIZE*CONV_SIZE;i++)
   {
       if(i>0 && (i%imageWidth==0))
       {
            fprintf(level4_1,"\n");

       }
     fprintf(level4_1, "%0.5f \t", *(hostMaskData+i));


   }
   fclose(level4_1);

   // Free conv_2_1 Memory
  free(hostMaskData);
   hipFree(deviceMaskData);
   hipFree(deviceBias);
   free(hostOutputImageData);
   hipFree(deviceOutputMaxPooledData3);
/*****************************conv_4_1 end************************************************/

/****************************conv_4_2 start**********************************************/
// Layer 4 (Convolution 128-> 128)
level = 8;
outputChannels = layers[level][0];
imageChannels = layers[level][1];


float * deviceOutputImageData_4_2;
  hostOutputImageData = (float *) malloc(sizeof(float)*imageWidth*imageHeight*outputChannels);
hostMaskData = (float *) malloc(sizeof(float)*outputChannels*imageChannels*CONV_SIZE*CONV_SIZE);
//hostOutputImageData = (float *) malloc(sizeof(float)*imageWidth*imageHeight*outputChannels

err = hipMalloc((void**)&deviceBias, imageWidth * imageHeight * imageChannels * sizeof(float));
 if (err != hipSuccess)
 {
     fprintf(stderr, "Failed to allocate deviceBias (error code %s)!\n", hipGetErrorString(err));
     exit(EXIT_FAILURE);
 }


err = hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * imageChannels*outputChannels* sizeof(float));
if (err != hipSuccess)
{
    fprintf(stderr, "Failed to allocate deviceMaskData(error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
}

err = hipMalloc((void **) &deviceOutputImageData_4_2, imageWidth * imageHeight *outputChannels* sizeof(float));
if (err != hipSuccess)
{
    fprintf(stderr, "Failed to allocate deviceOutputImageData(error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
}
readWeights(level,hostMaskData, bias);

printf("Copy input data from the host memory to the CUDA device level 4_2\n");
// Copy device bias
err = hipMemcpy(deviceBias, bias,
              sizeof(float)*layers[12][0],
             hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy input matrix from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
 // Copy device mask
  err = hipMemcpy(deviceMaskData,
             hostMaskData,
             outputChannels*imageChannels*CONV_SIZE*CONV_SIZE*sizeof(float),
             hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy mask matrix from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

convolution<<<dimGrid_4,dimBlock_4>>>(deviceOutputImageData_4_1, deviceMaskData, deviceOutputImageData_4_2, deviceBias,
                                   imageChannels, imageWidth, imageHeight,outputChannels);
hipMemcpy(hostOutputImageData,
         deviceOutputImageData_4_2,
         imageWidth * imageHeight * outputChannels * sizeof(float),
         hipMemcpyDeviceToHost);
hipDeviceSynchronize();
 // Program exits if the file pointer returns NULL.
 FILE *out4_2;
 if ((out4_2 = fopen("out4_2.txt","w")) == NULL){
     printf("Error! opening device file");
 exit(1);
 }
 for(int i=0;i<imageWidth*imageHeight*outputChannels;i++)
 {
     if(i>0 && (i%imageWidth==0))
     {
          fprintf(out4_2,"\n");

     }
   fprintf(out4_2, "%0.2f \t", *(hostOutputImageData+i));


 }
 fclose(out4_2);

 // Free conv_2_1 Memory
free(hostMaskData);
 hipFree(deviceMaskData);
 hipFree(deviceBias);
free(hostOutputImageData);
 hipFree(deviceOutputImageData_4_1);
/***************************conv_4_2 end************************************************/

/***************************conv_4_3 start************************************************/
// Layer 4 (Convolution 128-> 128)
level = 9;
outputChannels = layers[level][0];
imageChannels = layers[level][1];


float * deviceOutputImageData_4_3;

hostMaskData = (float *) malloc(sizeof(float)*outputChannels*imageChannels*CONV_SIZE*CONV_SIZE);
hostOutputImageData = (float *) malloc(sizeof(float)*imageWidth*imageHeight*outputChannels);



err = hipMalloc((void**)&deviceBias, imageWidth * imageHeight * imageChannels * sizeof(float));
 if (err != hipSuccess)
 {
     fprintf(stderr, "Failed to allocate deviceBias (error code %s)!\n", hipGetErrorString(err));
     exit(EXIT_FAILURE);
 }

err = hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * imageChannels*outputChannels* sizeof(float));
if (err != hipSuccess)
{
    fprintf(stderr, "Failed to allocate deviceMaskData(error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
}

err = hipMalloc((void **) &deviceOutputImageData_4_3, imageWidth * imageHeight *outputChannels* sizeof(float));
if (err != hipSuccess)
{
    fprintf(stderr, "Failed to allocate deviceOutputImageData(error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
}
readWeights(level,hostMaskData, bias);

printf("Copy input data from the host memory to the CUDA device level 4_3\n");
// Copy device bias
err = hipMemcpy(deviceBias, bias,
              sizeof(float)*layers[12][0],
             hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy input matrix from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
 // Copy device mask
  err = hipMemcpy(deviceMaskData,
             hostMaskData,
             outputChannels*imageChannels*CONV_SIZE*CONV_SIZE*sizeof(float),
             hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy mask matrix from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

convolution<<<dimGrid_4,dimBlock_4>>>(deviceOutputImageData_4_2, deviceMaskData, deviceOutputImageData_4_3, deviceBias,
                                   imageChannels, imageWidth, imageHeight,outputChannels);
hipMemcpy(hostOutputImageData,
         deviceOutputImageData_4_3,
         imageWidth * imageHeight * outputChannels * sizeof(float),
         hipMemcpyDeviceToHost);
hipDeviceSynchronize();
 // Program exits if the file pointer returns NULL.
 FILE *out4_3;
 if ((out4_3 = fopen("out4_2.txt","w")) == NULL){
     printf("Error! opening device file");
 exit(1);
 }
 for(int i=0;i<imageWidth*imageHeight*outputChannels;i++)
 {
     if(i>0 && (i%imageWidth==0))
     {
          fprintf(out4_3,"\n");

     }
   fprintf(out4_3, "%0.2f \t", *(hostOutputImageData+i));


 }
 fclose(out4_3);

 // Free conv_2_1 Memory
free(hostMaskData);
 hipFree(deviceMaskData);
 hipFree(deviceBias);
free(hostOutputImageData);
 hipFree(deviceOutputImageData_4_2);
 //hipFree(deviceOutputImageData_4_3);

/***************************conv_4_3 end************************************************/

/******************************max4 start**********************************************/
    float * deviceOutputMaxPooledData4;
     hostOutputMaxPooledData = (float *) malloc(sizeof(float)*imageWidth/2*imageHeight/2*outputChannels);
     //err = hipMalloc((void**) &deviceOutputMaxPooledData4, imageWidth/2 * imageHeight/2 * outputChannels * sizeof(float));
     err = hipMalloc((void**) &deviceOutputMaxPooledData4, imageWidth/2 * imageHeight/2 * outputChannels  * sizeof(float));
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to allocate deviceOutputMaxPooledData4 (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
     // image 28
     blockwidth = 4;
     number_blocks = imageWidth/blockwidth;
     dim3 dimGrid_m4(number_blocks,number_blocks,1);
     dim3 dimBlock_m4(blockwidth,blockwidth,1);
    maxpool<<<dimGrid_m4,dimBlock_m4>>>(deviceOutputImageData_4_3,deviceOutputMaxPooledData4 ,outputChannels, imageHeight, imageWidth, blockwidth);


  hipDeviceSynchronize();

  hipMemcpy(hostOutputMaxPooledData,
             deviceOutputMaxPooledData4,
             imageWidth/2 * imageHeight/2 * outputChannels * sizeof(float),
             hipMemcpyDeviceToHost);

     FILE *mp_4;

     if ((mp_4 = fopen("maxpooled4.txt","w")) == NULL){
         printf("Error! opening host file");

         // Program exits if the file pointer returns NULL.
         exit(1);
     }

      for(int i=0;i<imageWidth/2*imageHeight/2*outputChannels;i++)
         {
              if(i>0 && (i%(imageWidth/2 * outputChannels)==0))
                 fprintf(mp_4,"\n");

           fprintf(mp_4, "%0.2f \t", *(hostOutputMaxPooledData+i));
         }
  fclose(mp_4);
  //  hipFree(deviceOutputMaxPooledData);
    hipFree(deviceOutputImageData_4_3);

  //  hipFree(deviceInputMaxPool);
    free(hostOutputMaxPooledData);
/*****************************max4 end*************************************************/
/*****************************conv_5_1 start************************************************/
// Layer parameters
 imageWidth /= 2;
  imageHeight /= 2;

  // Layer 4 (Convolution 128 -> 256)
  level = 10;
  outputChannels = layers[level][0];
  imageChannels = layers[level][1];


  float * deviceOutputImageData_5_1;

  hostMaskData = (float *) malloc(sizeof(float)*outputChannels*imageChannels*CONV_SIZE*CONV_SIZE);
  hostOutputImageData = (float *) malloc(sizeof(float)*imageWidth*imageHeight*outputChannels);
  err = hipMalloc((void**)&deviceBias, imageWidth * imageHeight * imageChannels * sizeof(float));
   if (err != hipSuccess)
   {
       fprintf(stderr, "Failed to allocate deviceBias (error code %s)!\n", hipGetErrorString(err));
       exit(EXIT_FAILURE);
   }


  err = hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * imageChannels*outputChannels* sizeof(float));
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate deviceMaskData(error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

 err = hipMalloc((void **) &deviceOutputImageData_5_1, imageWidth * imageHeight *outputChannels* sizeof(float));
 if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate deviceOutputImageData(error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
  readWeights(level,hostMaskData, bias);

  printf("Copy input data from the host memory to the CUDA device level 5_1\n");
  // Copy device bias
  err = hipMemcpy(deviceBias, bias,
                sizeof(float)*layers[12][0],
               hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy input matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
   // Copy device mask
    err = hipMemcpy(deviceMaskData,
               hostMaskData,
               outputChannels*imageChannels*CONV_SIZE*CONV_SIZE*sizeof(float),
               hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy mask matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    dim3 dimGrid_5(((imageWidth-1)/TILE_WIDTH)+1, ((imageHeight-1)/TILE_WIDTH)+1,1);
    dim3 dimBlock_5(TILE_WIDTH, TILE_WIDTH, 1);
    convolution<<<dimGrid_5,dimBlock_5>>>(deviceOutputMaxPooledData4, deviceMaskData, deviceOutputImageData_5_1, deviceBias,
                                     imageChannels, imageWidth, imageHeight,outputChannels);
   err =   hipMemcpy(hostOutputImageData,
           deviceOutputImageData_5_1,
           imageWidth * imageHeight * outputChannels * sizeof(float),
           hipMemcpyDeviceToHost);

  if (err != hipSuccess)
   {
       fprintf(stderr, "Failed to copy 5.1 (error code %s)!\n", hipGetErrorString(err));
       exit(EXIT_FAILURE);
   }
  hipDeviceSynchronize();
   // Program exits if the file pointer returns NULL.
   FILE *out5_1;
   if ((out5_1 = fopen("out5_1.txt","w")) == NULL){
       printf("Error! opening device file");
   exit(1);
   }
   for(int i=0;i<imageWidth*imageHeight*outputChannels;i++)
   {
       if(i>0 && (i%imageWidth==0))
       {
            fprintf(out5_1,"\n");

       }
     fprintf(out5_1, "%0.2f \t", *(hostOutputImageData+i));


   }

   fclose(out5_1);

   FILE *level5_1;
   if ((level5_1 = fopen("level5_1.txt","w")) == NULL){
       printf("Error! opening device file");
   exit(1);
 }

   for(int i=0;i<outputChannels*imageChannels*CONV_SIZE*CONV_SIZE;i++)
   {
       if(i>0 && (i%imageWidth==0))
       {
            fprintf(level5_1,"\n");

       }
     fprintf(level5_1, "%0.5f \t", *(hostMaskData+i));


   }
   fclose(level5_1);

   // Free conv_2_1 Memory
  free(hostMaskData);
   hipFree(deviceMaskData);
   hipFree(deviceBias);
   free(hostOutputImageData);
   hipFree(deviceOutputMaxPooledData4);
/*****************************conv_5_1 end************************************************/
/****************************conv_5_2 start**********************************************/
// Layer 4 (Convolution 128-> 128)
level = 11;
outputChannels = layers[level][0];
imageChannels = layers[level][1];


float * deviceOutputImageData_5_2;
  hostOutputImageData = (float *) malloc(sizeof(float)*imageWidth*imageHeight*outputChannels);
hostMaskData = (float *) malloc(sizeof(float)*outputChannels*imageChannels*CONV_SIZE*CONV_SIZE);
//hostOutputImageData = (float *) malloc(sizeof(float)*imageWidth*imageHeight*outputChannels

err = hipMalloc((void**)&deviceBias, imageWidth * imageHeight * imageChannels * sizeof(float));
 if (err != hipSuccess)
 {
     fprintf(stderr, "Failed to allocate deviceBias (error code %s)!\n", hipGetErrorString(err));
     exit(EXIT_FAILURE);
 }


err = hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * imageChannels*outputChannels* sizeof(float));
if (err != hipSuccess)
{
    fprintf(stderr, "Failed to allocate deviceMaskData(error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
}

err = hipMalloc((void **) &deviceOutputImageData_5_2, imageWidth * imageHeight *outputChannels* sizeof(float));
if (err != hipSuccess)
{
    fprintf(stderr, "Failed to allocate deviceOutputImageData(error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
}
readWeights(level,hostMaskData, bias);

printf("Copy input data from the host memory to the CUDA device level 5_2\n");
// Copy device bias
err = hipMemcpy(deviceBias, bias,
              sizeof(float)*layers[12][0],
             hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy input matrix from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
 // Copy device mask
  err = hipMemcpy(deviceMaskData,
             hostMaskData,
             outputChannels*imageChannels*CONV_SIZE*CONV_SIZE*sizeof(float),
             hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy mask matrix from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

convolution<<<dimGrid_5,dimBlock_5>>>(deviceOutputImageData_5_1, deviceMaskData, deviceOutputImageData_5_2, deviceBias,
                                   imageChannels, imageWidth, imageHeight,outputChannels);
hipMemcpy(hostOutputImageData,
         deviceOutputImageData_5_2,
         imageWidth * imageHeight * outputChannels * sizeof(float),
         hipMemcpyDeviceToHost);
hipDeviceSynchronize();
 // Program exits if the file pointer returns NULL.
 FILE *out5_2;
 if ((out5_2 = fopen("out5_2.txt","w")) == NULL){
     printf("Error! opening device file");
 exit(1);
 }
 for(int i=0;i<imageWidth*imageHeight*outputChannels;i++)
 {
     if(i>0 && (i%imageWidth==0))
     {
          fprintf(out5_2,"\n");

     }
   fprintf(out5_2, "%0.2f \t", *(hostOutputImageData+i));


 }
 fclose(out5_2);

 // Free conv_2_1 Memory
free(hostMaskData);
 hipFree(deviceMaskData);
 hipFree(deviceBias);
free(hostOutputImageData);
 hipFree(deviceOutputImageData_5_1);
/***************************conv_4_2 end************************************************/

/***************************conv_4_3 start************************************************/
// Layer 5 (Convolution 128-> 128)
level = 12;
outputChannels = layers[level][0];
imageChannels = layers[level][1];


float * deviceOutputImageData_5_3;

hostMaskData = (float *) malloc(sizeof(float)*outputChannels*imageChannels*CONV_SIZE*CONV_SIZE);
hostOutputImageData = (float *) malloc(sizeof(float)*imageWidth*imageHeight*outputChannels);



err = hipMalloc((void**)&deviceBias, imageWidth * imageHeight * imageChannels * sizeof(float));
 if (err != hipSuccess)
 {
     fprintf(stderr, "Failed to allocate deviceBias (error code %s)!\n", hipGetErrorString(err));
     exit(EXIT_FAILURE);
 }

err = hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * imageChannels*outputChannels* sizeof(float));
if (err != hipSuccess)
{
    fprintf(stderr, "Failed to allocate deviceMaskData(error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
}

err = hipMalloc((void **) &deviceOutputImageData_5_3, imageWidth * imageHeight *outputChannels* sizeof(float));
if (err != hipSuccess)
{
    fprintf(stderr, "Failed to allocate deviceOutputImageData(error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
}
readWeights(level,hostMaskData, bias);

printf("Copy input data from the host memory to the CUDA device level 5_3\n");
// Copy device bias
err = hipMemcpy(deviceBias, bias,
              sizeof(float)*layers[12][0],
             hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy input matrix from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
 // Copy device mask
  err = hipMemcpy(deviceMaskData,
             hostMaskData,
             outputChannels*imageChannels*CONV_SIZE*CONV_SIZE*sizeof(float),
             hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy mask matrix from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

convolution<<<dimGrid_5,dimBlock_5>>>(deviceOutputImageData_5_2, deviceMaskData, deviceOutputImageData_5_3, deviceBias,
                                   imageChannels, imageWidth, imageHeight,outputChannels);
hipMemcpy(hostOutputImageData,
         deviceOutputImageData_5_3,
         imageWidth * imageHeight * outputChannels * sizeof(float),
         hipMemcpyDeviceToHost);
hipDeviceSynchronize();
 // Program exits if the file pointer returns NULL.
 FILE *out5_3;
 if ((out5_3 = fopen("out5_3.txt","w")) == NULL){
     printf("Error! opening device file");
 exit(1);
 }
 for(int i=0;i<imageWidth*imageHeight*outputChannels;i++)
 {
     if(i>0 && (i%imageWidth==0))
     {
          fprintf(out5_3,"\n");

     }
   fprintf(out5_3, "%0.2f \t", *(hostOutputImageData+i));


 }
 fclose(out5_3);

 // Free conv_2_1 Memory
free(hostMaskData);
 hipFree(deviceMaskData);
 hipFree(deviceBias);
free(hostOutputImageData);
 hipFree(deviceOutputImageData_5_2);


/***************************conv_5_3 end************************************************/
/******************************max5 start**********************************************/
    float * deviceOutputMaxPooledData5;
     hostOutputMaxPooledData = (float *) malloc(sizeof(float)*imageWidth/2*imageHeight/2*outputChannels);
     //err = hipMalloc((void**) &deviceOutputMaxPooledData4, imageWidth/2 * imageHeight/2 * outputChannels * sizeof(float));
     err = hipMalloc((void**) &deviceOutputMaxPooledData5, imageWidth/2 * imageHeight/2 * outputChannels  * sizeof(float));
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to allocate deviceOutputMaxPooledData5 (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
     // image 28
     blockwidth = 2;
     number_blocks = imageWidth/blockwidth;
     dim3 dimGrid_m5(number_blocks,number_blocks,1);
     dim3 dimBlock_m5(blockwidth,blockwidth,1);
    maxpool<<<dimGrid_m5,dimBlock_m5>>>(deviceOutputImageData_5_3,deviceOutputMaxPooledData5 ,outputChannels, imageHeight, imageWidth, blockwidth);


  hipDeviceSynchronize();

  hipMemcpy(hostOutputMaxPooledData,
             deviceOutputMaxPooledData4,
             imageWidth/2 * imageHeight/2 * outputChannels * sizeof(float),
             hipMemcpyDeviceToHost);

     FILE *mp_5;

     if ((mp_5 = fopen("maxpooled5.txt","w")) == NULL){
         printf("Error! opening host file");

         // Program exits if the file pointer returns NULL.
         exit(1);
     }

      for(int i=0;i<imageWidth/2*imageHeight/2*outputChannels;i++)
         {
              if(i>0 && (i%(imageWidth/2 * outputChannels)==0))
                 fprintf(mp_5,"\n");

           fprintf(mp_5, "%0.2f \t", *(hostOutputMaxPooledData+i));
         }
  fclose(mp_5);
  //  hipFree(deviceOutputMaxPooledData);
    hipFree(deviceOutputImageData_5_3);

  //  hipFree(deviceInputMaxPool);
    free(hostOutputMaxPooledData);
/*****************************max5 end*************************************************/
/*****************************dense_1_1 start************************************************/
// Layer parameters
 imageWidth /= 2;
  imageHeight /= 2;

  // Layer 4 (Convolution 128 -> 256)
  level = 0;
  int input = dense[level][0];
  int output = dense[level][1];


  float * deviceOutputImageDataDense_1_1;

  //hostMaskData = (float *) malloc(sizeof(float)*output*input);
  hostOutputImageData = (float *) malloc(sizeof(float)*output);
  err = hipMalloc((void**)&deviceBias, output * sizeof(float));
   if (err != hipSuccess)
   {
       fprintf(stderr, "Failed to allocate deviceBias (error code %s)!\n", hipGetErrorString(err));
       exit(EXIT_FAILURE);
   }


  err = hipMalloc((void **) &deviceMaskData, output*input*sizeof(float));
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate deviceMaskData(error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

 err = hipMalloc((void **) &deviceOutputImageDataDense_1_1,output* sizeof(float));
 if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate deviceOutputImageData(error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
  //dense_weights(level,hostMaskData, biasDense);

  printf("Copy input data from the host memory to the CUDA device level FC 1_1\n");
  // Copy device bias
  err = hipMemcpy(deviceBias, bias_1,
                sizeof(float)*dense[0][1],
               hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy input matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
   // Copy device mask

    err = hipMemcpy(deviceMaskData,
               dense_1,
               output*input*sizeof(float),
               hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy mask matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    dim3 dimGrid_fc1_1(1,1,1);
    dim3 dimBlock_fc1_1(imageWidth, imageWidth, 1);
  //  fully1(float *I, const float* __restrict__ M, float *P,int channels,int outputChannels)
//  __global__ void fully1(float *I, const float* __restrict__ M, float *P,int channels,int outputChannels)
//int channels, int width, int height,int outputChannels)

  fully1<<<dimGrid_fc1_1,dimBlock_fc1_1>>>(deviceOutputMaxPooledData5, deviceMaskData, deviceOutputImageDataDense_1_1,
                                   512,output,deviceBias);
   err =   hipMemcpy(hostOutputImageData,
           deviceOutputImageDataDense_1_1,
           output* sizeof(float),
           hipMemcpyDeviceToHost);

  if (err != hipSuccess)
   {
       fprintf(stderr, "Failed to copy f1_1 (error code %s)!\n", hipGetErrorString(err));
       exit(EXIT_FAILURE);
   }
  hipDeviceSynchronize();
   // Program exits if the file pointer returns NULL.
   FILE *outf1_1;
   if ((outf1_1 = fopen("outf1_1.txt","w")) == NULL){
       printf("Error! opening device file");
   exit(1);
   }
   for(int i=0;i<output;i++)
   {

     fprintf(outf1_1, "%0.2f \t", *(hostOutputImageData+i));


   }

   fclose(outf1_1);

   FILE *levelf1_1;
   if ((levelf1_1 = fopen("levelf1_1.txt","w")) == NULL){
       printf("Error! opening device file");
   exit(1);
 }

   for(int i=0;i<output;i++)
   {


     fprintf(levelf1_1, "%0.5f \t", *(hostMaskData+i));


   }
   fclose(levelf1_1);

   // Free conv_2_1 Memory
//  free(hostMaskData);
   hipFree(deviceMaskData);
   hipFree(deviceBias);
   free(hostOutputImageData);
   hipFree(deviceOutputMaxPooledData5);
/*****************************conv_5_1 end************************************************/
/*****************************dense_1_2 start************************************************/


  // Layer 4 (Convolution 128 -> 256)
  level = 1;
   input = dense[level][0];
   output = dense[level][1];


  float * deviceOutputImageDataDense_1_2;

//  hostMaskData = (float *) malloc(sizeof(float)*output*input);
  hostOutputImageData = (float *) malloc(sizeof(float)*output);
  err = hipMalloc((void**)&deviceBias, output * sizeof(float));
   if (err != hipSuccess)
   {
       fprintf(stderr, "Failed to allocate deviceBias (error code %s)!\n", hipGetErrorString(err));
       exit(EXIT_FAILURE);
   }


  err = hipMalloc((void **) &deviceMaskData, output*input*sizeof(float));
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate deviceMaskData(error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

 err = hipMalloc((void **) &deviceOutputImageDataDense_1_2,output* sizeof(float));
 if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate deviceOutputImageData(error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
  //dense_weights(level,hostMaskData, biasDense);

  printf("Copy input data from the host memory to the CUDA device level FC 1_2\n");
  // Copy device bias
  err = hipMemcpy(deviceBias, bias_2,
                sizeof(float)*dense[1][1],
               hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy input matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
   // Copy device mask
    err = hipMemcpy(deviceMaskData,
               dense_2,
               output*input*sizeof(float),
               hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy mask matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    dim3 dimGrid_fc1_2(1,1,1);
    dim3 dimBlock_fc1_2(32, 32, 1);
  //  fully1(float *I, const float* __restrict__ M, float *P,int channels,int outputChannels)
    fully2<<<dimGrid_fc1_2,dimBlock_fc1_2>>>(deviceOutputImageDataDense_1_1, deviceMaskData, deviceOutputImageDataDense_1_2,
                                   4096,output, deviceBias);
   err =   hipMemcpy(hostOutputImageData,
           deviceOutputImageDataDense_1_2,
           output* sizeof(float),
           hipMemcpyDeviceToHost);

  if (err != hipSuccess)
   {
       fprintf(stderr, "Failed to copy FC 1.2 (error code %s)!\n", hipGetErrorString(err));
       exit(EXIT_FAILURE);
   }
  hipDeviceSynchronize();
   // Program exits if the file pointer returns NULL.
   FILE *outf1_2;
   if ((outf1_2 = fopen("outf1_2.txt","w")) == NULL){
       printf("Error! opening device file");
   exit(1);
   }
   for(int i=0;i<output;i++)
   {

     fprintf(outf1_2, "%0.2f \t", *(hostOutputImageData+i));


   }

   fclose(outf1_2);

   FILE *levelf1_2;
   if ((levelf1_2 = fopen("levelf1_1.txt","w")) == NULL){
       printf("Error! opening device file");
   exit(1);
 }

   for(int i=0;i<output;i++)
   {


     fprintf(levelf1_2, "%0.5f \t", *(hostMaskData+i));


   }
   fclose(levelf1_2);
    hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&time_taken,start,stop);
   int seconds = (time_taken/1000);
   int milli_seconds = ((int)time_taken%1000);
   printf("total elapsed time taken seconds is: %d milli-seconds: %d \n",seconds, milli_seconds );

   // Free conv_2_1 Memory
//  free(hostMaskData);
   hipFree(deviceMaskData);
   hipFree(deviceBias);
   free(hostOutputImageData);
   hipFree(deviceOutputImageDataDense_1_1);
/*****************************conv_5_1 end************************************************/

/*****************************dense_1_3 start************************************************/


  // Layer 4 (Convolution 128 -> 256)
  level = 2;
   input = dense[level][0];
   output = dense[level][1];


  float * deviceOutputImageDataDense_1_3;

  hostMaskData = (float *) malloc(sizeof(float)*output*input);
  hostOutputImageData = (float *) malloc(sizeof(float)*output);
  err = hipMalloc((void**)&deviceBias, output * sizeof(float));
   if (err != hipSuccess)
   {
       fprintf(stderr, "Failed to allocate deviceBias (error code %s)!\n", hipGetErrorString(err));
       exit(EXIT_FAILURE);
   }


  err = hipMalloc((void **) &deviceMaskData, output*input*sizeof(float));
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate deviceMaskData(error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

 err = hipMalloc((void **) &deviceOutputImageDataDense_1_3,output* sizeof(float));
 if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate deviceOutputImageData(error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
  //dense_weights(level,hostMaskData, biasDense);

  printf("Copy input data from the host memory to the CUDA device level FC 1_3\n");
  // Copy device bias
  err = hipMemcpy(deviceBias, bias_3,
                sizeof(float)*dense[2][1],
               hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy input matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
   // Copy device mask

    err = hipMemcpy(deviceMaskData,
               dense_3,
               output*input*sizeof(float),
               hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy mask matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    dim3 dimGrid_fc1_3(1,1,1);
    dim3 dimBlock_fc1_3(32, 32, 1);
  //  fully1(float *I, const float* __restrict__ M, float *P,int channels,int outputChannels)
    fully3<<<dimGrid_fc1_3,dimBlock_fc1_3>>>(deviceOutputImageDataDense_1_2, deviceMaskData, deviceOutputImageDataDense_1_3,
                                   4096,output,deviceBias);


   err =   hipMemcpy(hostOutputImageData,
           deviceOutputImageDataDense_1_3,
           output* sizeof(float),
           hipMemcpyDeviceToHost);

  if (err != hipSuccess)
   {
       fprintf(stderr, "Failed to copy FC 1_3 (error code %s)!\n", hipGetErrorString(err));
       exit(EXIT_FAILURE);
   }
   hipDeviceSynchronize();
   // Program exits if the file pointer returns NULL.
   FILE *outf1_3;
   if ((outf1_3 = fopen("outf1_3.txt","w")) == NULL){
       printf("Error! opening device file");
   exit(1);
   }
   for(int i=0;i<output;i++)
   {

     fprintf(outf1_3, "%0.2f \t", *(hostOutputImageData+i));


   }

   fclose(outf1_3);


  softmax(hostOutputImageData, 1000);

  FILE *soft;
  if ((soft = fopen("softmax.txt","w")) == NULL){
      printf("Error! opening device file");
  exit(1);
}
 float max =0;
 int j=0;
  for(int i=0;i<1000;i++)
  {

    if(*(hostOutputImageData+i) > max)
    {
      max = *(hostOutputImageData+i);
      j =i;
    }
//    fprintf(soft, "%0.5f \t", *(hostOutputImageData+i));
  }
  fprintf(soft, "class:%d at:%d \n", max,j);
  fclose(soft);


  // Free conv_2_1 Memory
 free(hostMaskData);
  hipFree(deviceMaskData);
  hipFree(deviceBias);
  free(hostOutputImageData);
  hipFree(deviceOutputImageDataDense_1_2);
  hipFree(deviceOutputImageDataDense_1_3);
  free(dense_1);
  free(dense_2);
  free(dense_3);
  free(bias_1);
  free(bias_2);
  free(bias_3);

/*****************************cdense1_3 end************************************************/



  printf("\n Number of Threads Per Block created in code: %d",TILE_WIDTH*TILE_WIDTH);
  printf("\n Number of Blocks Created :%d",(((imageWidth-1)/TILE_WIDTH)+1)*(((imageWidth-1)/TILE_WIDTH)+1));
  printf("No Error");
  return 0;
}

float convolution_2D_OnHost(float * N,float * M,int width, int height,int i,int j,int imageChannels, int outputChannels)
{
 float Pvalue=0.0;
 int N_start_point_i = i  - (Mask_width/2);
 int N_start_point_j = j  - (Mask_height/2);
 for(int j = 0; j<imageChannels; j++)
 {

       for(int k=0;k<Mask_width;k++)
       {
          for(int l=0;l<Mask_height;l++)
          {
             if(((N_start_point_i+k)>=0) && ((N_start_point_i+k)<width)&&((N_start_point_j+l)>=0)&&((N_start_point_j+l)<height))
             {
                 Pvalue +=N[((N_start_point_i+k)*width+(N_start_point_j+l))*imageChannels + j] *M[ (outputChannels*Mask_width*Mask_width*imageChannels) + (j*Mask_width*Mask_width) + (k*Mask_width)+l];
             }
         }
       }

}
// return(clamp(Pvalue));

 return((Pvalue));
}

/***/
